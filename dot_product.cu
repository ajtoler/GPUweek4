
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// Use atomicAdd	
    int idx =  threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) atomicAdd(C, A[idx] * B[idx]);
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}


	*h_C = 0;


	// Allocate device memory 
	hipMalloc(&d_A, DSIZE*sizeof(int));
	hipMalloc(&d_B, DSIZE*sizeof(int));
	hipMalloc(&d_C, DSIZE*sizeof(int));

	// Check memory allocation for errors
	cudaCheckErrors();

	// Copy the matrices on GPU
	hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, DSIZE*sizeof(int), hipMemcpyHostToDevice);

	// Check memory copy for errors
    cudaCheckErrors();

	// Define block/grid dimensions and launch kernel
	int block = BLOCK_SIZE;
	int grid((DSIZE + block - 1) / block);
	dot_product<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

	// Copy results back to host
	hipMemcpy(h_C, d_C, DSIZE*sizeof(int), hipMemcpyDeviceToHost);

    // Check copy for errors
    cudaCheckErrors();

	// Verify result
    // for (int i = 0; i < DSIZE; i++) printf("A[%d]: %d, B[%d]: %d\n", i, h_A[i], i, h_B[i]);
	printf("Dot Product C: %d\n", *h_C);

	// Free allocated memory
	free(h_A);
    free(h_B);
	free(h_C);
    hipFree(d_A);
    hipFree(d_B);
	hipFree(d_C);
	
	return 0;

}